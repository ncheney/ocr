#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include <fn/hmm/gpu_update.h>
#include <fn/hmm/layout.h>
#include <fn/hmm/output.h>


/*! Allocate memory for, and copy an HMM network to, a GPU.
 */
void* fn::hmm::gpu_alloc(hmm_header* hdr, std::size_t size) {
	void* dmem=0;
	hipMalloc(&dmem, size);
	assert(dmem);
	hipMemcpy(dmem, hdr, size, hipMemcpyHostToDevice);
	return dmem;
}


/*! Deallocate memory on a GPU.
 */
void fn::hmm::gpu_free(void* dmem) {
	assert(dmem);
	hipFree(dmem);
}


/*! Generate a park-miller psuedorandom number.
 
 Derived from Langdon, GECCO'09.
 */
__device__ int parkmiller_rand(unsigned int seed, int cycles) {
	float const a=16807;
	float const m=2147483647;
	float const reciprocal_m = 1.0/m;
	unsigned int data=seed+blockDim.x*blockIdx.x + threadIdx.x;
	unsigned int result=0;
	
	for(int i=1; i<=cycles; ++i) {
		float temp = data * a;
		result = (int) (temp - m * floor(temp * reciprocal_m));
		data = result;
	}
	
	return result;
}


/*! Generate a psuedorandom number via linear congruential generation.
 */
__device__ int lcg_rand(int a) {
	return ((long)a * 279470273UL) % 4294967291UL;
}


/*! Update the GPU's copy of the HMM.
 */
__global__ void update(void* dmem, int seed) {
	using namespace fn::hmm;
	hmm_header* hdr=reinterpret_cast<hmm_header*>(dmem);
	node_header* ndr=reinterpret_cast<node_header*>(hdr->node_ptr(threadIdx.x));
	
	switch(ndr->start_codon[0]) {
		case 42: { // probabilistic node
			probabilistic_output(hdr,ndr,parkmiller_rand(seed,1));
			break;
		}
		case 43: { // deterministic node
			deterministic_output(hdr,ndr);
			break;
		}
		default:
			break;
	}
}


/*! Update this network on the GPU.
 
 In this case, we're copying the whole header to the device before the update,
 running the update, and then copying the header back to the host.  We can't get
 away with just copying the state vectors, as we need the offsets for t and t-1.
 
 So, in order to reduce the number of memcpys, we copy a few extra bytes and grab
 the whole header.
 */		
void fn::hmm::gpu_update(hmm_header* hdr, void* dmem, int seed) {
	// copy the header to the device:
	hipMemcpy(dmem, hdr, hdr->header_size(), hipMemcpyHostToDevice);
	
	update<<<1,hdr->nnodes>>>(dmem, seed);
	
	// copy the header from the device:
	hipMemcpy(hdr, dmem, hdr->header_size(), hipMemcpyDeviceToHost);
}
